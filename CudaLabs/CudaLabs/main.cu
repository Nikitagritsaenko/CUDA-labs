#include "hip/hip_runtime.h"
﻿#include <new>
#include <algorithm>
#include <random>
#include <iostream>
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""

#define DIM 1000
#define MATRIX_MIN_VALUE -100
#define MATRIX_MAX_VALUE 100
#define BLOCK_SIZE 16

double* generateRandomMatrix(size_t dim)
{
	std::random_device device;
	std::mt19937 generator(device());
	std::uniform_real_distribution<double> distribution(MATRIX_MIN_VALUE, MATRIX_MAX_VALUE);

	size_t elementsTotal = dim * dim;
	double* randomMatrix = new double[elementsTotal];

	for (size_t i = 0; i < elementsTotal; ++i) 
	{
		randomMatrix[i] = distribution(generator);
	}

	return randomMatrix;
}

float cpuMatrixMultiplication(double* A, double* B, double* C, size_t n)
{
	LARGE_INTEGER frequency;
	QueryPerformanceFrequency(&frequency);

	LARGE_INTEGER start;
	QueryPerformanceCounter(&start);

	for (size_t i = 0; i < n; ++i)
	{
		for (size_t j = 0; j < n; ++j)
		{
			for (size_t k = 0; k < n; ++k)
			{
				C[i * n + j] += A[i * n + k] * B[k * n + j];
			}
		}
	}

	LARGE_INTEGER end;
	QueryPerformanceCounter(&end);

	return (float)(end.QuadPart - start.QuadPart) / frequency.QuadPart;
}

__global__ void matrixMul(double* A, double* B, double* C, size_t n)
{
	size_t row = blockIdx.y * blockDim.y + threadIdx.y;
	size_t column = blockIdx.x * blockDim.x + threadIdx.x;

	double cellValue = 0;
	if (row < n && column < n)
	{
		for (int i = 0; i < n; ++i)
		{
			cellValue += A[row * n + i] * B[i * n + column];
		}
		C[row * n + column] = cellValue;
	}
}

float gpuMatrixMultiplication(double* A, double* B, double* C, size_t n)
{
	double* dA;
	double* dB;
	double* dC;
	size_t matrixSizeInBytes = n * n * sizeof(double);

	hipMalloc(&dA, matrixSizeInBytes);
	hipMalloc(&dB, matrixSizeInBytes);
	hipMalloc(&dC, matrixSizeInBytes);

	hipMemcpy(dA, A, matrixSizeInBytes, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, matrixSizeInBytes, hipMemcpyHostToDevice);

	int gridSize = n / BLOCK_SIZE + 1;

	dim3 grid(gridSize, gridSize);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	hipEvent_t start, end;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start, 0);

	matrixMul << <grid, threads >> > (dA, dB, dC, n);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);

	hipEventDestroy(start);
	hipEventDestroy(end);

	hipMemcpy(C, dC, matrixSizeInBytes, hipMemcpyDeviceToHost);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return time / 1000.0f;
}

double getMaximumDeviation(double* A, double* B, size_t n)
{
	double maxDeviation = 0.0;

	for (size_t i = 0; i < n * n; ++i) 
	{
		maxDeviation = std::max(maxDeviation, std::abs(A[i] - B[i]));
	}

	return maxDeviation;
}

int main()
{
	size_t n = DIM;

	double* mA = generateRandomMatrix(n);
	double* mB = generateRandomMatrix(n);

	double* resultCPU = new double[n * n];
	std::fill_n(resultCPU, n * n, 0);

	float cpuTime = cpuMatrixMultiplication(mA, mB, resultCPU, n);
	
	double* resultGPU = new double[n * n];
	std::fill_n(resultGPU, n * n, 0);

	float gpuTime = gpuMatrixMultiplication(mA, mB, resultGPU, n);

	double maxDeviation = getMaximumDeviation(resultCPU, resultGPU, n);

	std::cout << "CPU time = " << cpuTime << std::endl;
	std::cout << "GPU time = " << gpuTime << std::endl;
	std::cout << "Maximum deviation = " << maxDeviation << std::endl;

	delete[] mA;
	delete[] mB;
	delete[] resultCPU;
	delete[] resultGPU;

	return 0;
}